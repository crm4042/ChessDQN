#include "hip/hip_runtime.h"
#include "FeedForward.cuh"


// The feedforward algorithm propogates the inputs given forward.
// Because these are dependant on the layer before it, the threads must be called
// layer by layer. Furthermore, this can only be parallelized with one thread per
// output because it is an nx1 input and has a race condition in that the
// weights/biases can overwrite one another if called to update the same neuron
// in parallel.


/**
  *	The kernal for the feedforward algorithm
  *	Parameter nn: the neural network for the feedforward alforithm
  *	Parameter layer: the layer that is being evaluated
  *	Parameter outputs: the output matrix
  *	Return: nothing
  */

__global__ 
void feedForwardKernel(NeuralNet* nn, int layer, double*** outputs){

	// Loops through the output neurons
	for(int neuron2 = threadIdx.x + (blockIdx.x * blockDim.x); 
			neuron2 < nn->neurons[layer]; 
			neuron2 += blockDim.x*gridDim.x){

		// Sets the initial output to 0
		(*outputs)[layer][neuron2] = 0;

		// Loops through the input neurons and multiplies the weights * the inputs
		for(int neuron1 = 0; neuron1 < nn->neurons[layer-1]; neuron1++){
			(*outputs)[layer][neuron2] += 
				(nn->weights[layer-1][neuron1][neuron2] * 
				(*outputs)[layer-1][neuron1]);
		}

		// Adds the bias
		(*outputs)[layer][neuron2]+=nn->biases[layer-1][neuron2];
		switch(nn->activations[layer-1][neuron2]){
			case BINARY_STEP:
				if((*outputs)[layer][neuron2] <=0){
					(*outputs)[layer][neuron2] = 0;
				}
				else{
					(*outputs)[layer][neuron2] = 1;
				}
				break;
			case LOGISTIC:
				(*outputs)[layer][neuron2] = 
					1/(1+exp(-1*(*outputs)[layer][neuron2]));
				break;
			case TANH:
				(*outputs)[layer][neuron2] = 
					tanh((*outputs)[layer][neuron2]);
				break;
			case RELU:
				if((*outputs)[layer][neuron2] < 0){
					(*outputs)[layer][neuron2] = 0;
				}
				break;
			case LEAKYRELU:
				if((*outputs)[layer][neuron2] < 0){
					(*outputs)[layer][neuron2] *= .01;
				}
				break;
		}
	}
}

/**
  *	Feeds the inputs forward through the neural network.
  *	Parameter nn: the neural network to feed the inputs through
  *	Parameter outputs: a reference to the output matrix
  *	Parameter inputs: the array of inputs
  *	Returns: nothing
  */

void feedForward(NeuralNet* nn, double*** outputs, double* inputs){

	// Loops through the layers
	for(int layer = 0; layer < nn->layers; layer++){

		// If it is the input layer
		if(layer == 0){

			// Sets the input layer to the inputs
			for(int input = 0; input < nn->neurons[layer]; input++){
				(*outputs)[layer][input] = inputs[input];
			}
		}

		else{
			// Calls the feedforward kernel
			feedForwardKernel<<<NUMBLOCKS, BLOCKSIZE>>>(nn, layer, 
					outputs);

			hipDeviceSynchronize();
		}
	}
}

/**
  *	Makes the output matrix
  *	Parameter nn: the neural net to make the output matrix from
  *	Parameter numOutputs: the number of outputs to create
  *	Return: the output matrix created
  */

double*** makeOutputs(NeuralNet* nn, int numOutputs){
	double*** outputs;
	hipMallocManaged(&outputs, numOutputs * sizeof(double**));
	
	for(int output = 0; output < numOutputs; output++){
		hipMallocManaged(&outputs[output], 
				nn->layers * sizeof(double*));

		for(int layer=0; layer < nn->layers; layer++){
			hipMallocManaged(&outputs[output][layer], 
					nn->neurons[layer] * sizeof(double));
		}
	}
	return outputs;
}

/**
  *	Frees the output matrix
  *	Parameter nn: the neural network
  *	Parameter outputs: the outout matrix to free
  *	Parameter numOutputs: the number of outputs in the matrix
  *	Returns: nothing
  */

void freeOutputs(NeuralNet* nn, double*** outputs, int numOutputs){
	for(int output=0; output < numOutputs; output++){
		for(int layer=0; layer < nn->layers; layer++){
			hipFree(outputs[output][layer]);
		}
		hipFree(outputs[output]);
	}
	hipFree(outputs);
}

