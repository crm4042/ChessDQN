#include "NeuralNetwork.cuh"

/**
  *	Creates a neural network with the specified number of layers
  *	and neurons
  *	Parameter layers: the number of layers in the neural network
  *	Parameter neurons: an array with the number of neurons for each layer
  *	Returns: a NeuralNet with the specified layers/neurons
  */

NeuralNet* createNeuralNet(int layers, int* neurons, activation** activations){
	// Seeds the randomizer
	srand(time(NULL));

	// Creates the neural net
	NeuralNet* nn;
	hipMallocManaged(&nn, sizeof(NeuralNet));
	
	// Sets the attributes of the neural net
	nn->layers = layers;
	hipMallocManaged(&nn->neurons, layers*sizeof(int));
	for(int layer=0; layer<layers; layer++){
		nn->neurons[layer] = neurons[layer];
	}

	// Allocates memory for the activation function enum
	hipMallocManaged(&nn->activations, 
			(layers - 1) * sizeof(activation*));

	// Allocates memory for the weights/biases and assigns random values
	hipMallocManaged(&nn->biases, (layers - 1) * sizeof(double*));
	hipMallocManaged(&nn->weights, (layers - 1) * sizeof(double**));
	
	for(int layer = 0; layer < layers-1; layer++){

		hipMallocManaged(&nn->activations[layer], 
				neurons[layer + 1] *sizeof(activation));

		hipMallocManaged(&nn->biases[layer], 
				neurons[layer+1] * sizeof(double));
		hipMallocManaged(&nn->weights[layer], 
				neurons[layer] * sizeof(double*));

		for(int neuron1 = 0; neuron1 < neurons[layer+1]; neuron1++){
			nn->biases[layer][neuron1] = double(rand())/RAND_MAX;
			nn->activations[layer][neuron1] = 
				activations[layer][neuron1];
		}

		for(int neuron1 = 0; neuron1 < neurons[layer]; neuron1++){
			hipMallocManaged(&nn->weights[layer][neuron1], 
					neurons[layer+1] * sizeof(double));
			
			for(int neuron2 = 0; neuron2 < neurons[layer+1]; 
					neuron2++){
				nn->weights[layer][neuron1][neuron2] = 
					double(rand())/RAND_MAX;
			}
		}
	}

	return nn;
}

void freeNeuralNet(NeuralNet* nn){
	for(int layer = 0; layer < nn->layers-1; layer++){
		for(int neuron1=0; neuron1 < nn->neurons[layer]; neuron1++){
			hipFree(nn->weights[layer][neuron1]);
		}
		hipFree(nn->activations[layer]);
		hipFree(nn->biases[layer]);
		hipFree(nn->weights[layer]);
	}
	hipFree(nn->neurons);
	hipFree(nn->activations);
	hipFree(nn->biases);
	hipFree(nn->weights);
	hipFree(nn);
}

void printNeuralNet(NeuralNet* nn){
	printf("Layers = %d\n", nn->layers);
	for(int layer = 0; layer < nn->layers; layer++){

		printf("\nLayer %d Neurons %d\n", layer, nn->neurons[layer]);

		// Prints the biases
		if(layer != 0){
			printf("Activations for this layer:\n");
			for(int neuron1 = 0; neuron1 < nn->neurons[layer]; 
					neuron1++){
				printf("%d\t", nn->activations[layer-1][neuron1]);
			}
			printf("\n");
			printf("Biases for this layer:\n");
			for(int neuron1 = 0; neuron1 < nn->neurons[layer]; 
					neuron1++){
				printf("%f\t", nn->biases[layer-1][neuron1]);
			}
			printf("\n");
		}
		else{
			printf("No biases in this layer\n");
		}

		// Prints the weights
		if(layer != nn->layers - 1){
			printf("Weights for this layer:\n");
			for(int neuron1 = 0; neuron1 < nn->neurons[layer];
					neuron1++){
				for(int neuron2 = 0; 
						neuron2 < nn->neurons[layer+1];
						neuron2++){
					printf("%f\t", nn->weights[layer]\
							[neuron1][neuron2]);
				}
				printf("\n");
			}
		}
	}
}

