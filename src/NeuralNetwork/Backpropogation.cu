#include "hip/hip_runtime.h"
#include "Backpropogation.cuh"
#include "NeuralNet.cuh"

/**
  *	Gets the errors associated with each neuron's output
  *	Parameter nn: the neural network to get the error for
  *	Parameter errors: the matrix to put the errors in
  *	Parameter layer: the layer to get the error for
  *	Return: nothing
  */

__global__
void getErrorKernel(NeuralNet* nn, double** errors, double** outputs, double* expected, int layer){

	for(int neuron1 = threadIdx.x + (blockDim.x * blockIdx.x); neuron1 < nn ->neurons[layer]; neuron1 += blockDim.x*gridDim.x){

		// The last layer's error
		if(layer == nn->layers - 1){
			//printf("actual - expected = %f - %f = %f\n", outputs[nn->layers - 1][neuron1], expected[neuron1], outputs[nn->layers - 1][neuron1]-expected[neuron1]);

			errors[layer][neuron1] = outputs[nn->layers - 1][neuron1] - expected[neuron1];
		}
	
		// The remaining layers' errors
		else{

			// Zeros the error
			errors[layer][neuron1] = 0;
	
			// Uses the derivative of the outputs with respect to the inputs
			for(int neuron2 = 0; neuron2 < nn->neurons[layer+1]; neuron2++){
				switch(nn->activations[layer][neuron2]){
					case LINEAR:
						errors[layer][neuron1] += 
						       (errors[layer + 1][neuron2] * 
							nn -> weights[layer][neuron1][neuron2]);
						break;
	
					case BINARY_STEP:
						break;
	
					case LOGISTIC:
						errors[layer][neuron1] += 
							(errors[layer + 1][neuron2] * 
							 nn -> weights[layer][neuron1][neuron2] * 
							 outputs[layer + 1][neuron2] * 
							 (1 - outputs[layer + 1][neuron2]));
						break;
	
					case TANH:
						errors[layer][neuron1] += 
							(errors[layer + 1][neuron2] * 
							 nn -> weights[layer][neuron1][neuron2] * 
							 (1 - (outputs[layer + 1][neuron2] * 
							       outputs[layer + 1][neuron2])));
						break;
	
					case RELU:
						if(outputs[layer + 1][neuron2] > 0){
							errors[layer][neuron1] += 
								(errors[layer + 1][neuron2] * 
							 	nn -> weights[layer][neuron1][neuron2]);
						}
						break;
	
					case LEAKYRELU:
						if(outputs[layer + 1][neuron2] < 0){
							errors[layer][neuron1] += 
								(errors[layer + 1][neuron2] * 
								 nn -> weights[layer][neuron1][neuron2] * .01);
						}
	
						else{
							errors[layer][neuron1] += 
								(errors[layer + 1][neuron2] * 
								 nn -> weights[layer][neuron1][neuron2]);
						}
						break;
				}
			}
		}
	}
}

/**
  *	Backpropogates the weights in the neural net
  *	Parameter nn: the neural network to backpropogate
  *	Parameter layer: the layer that is being backpropogated
  *	Returns: nothing
  */

__global__
void backpropogationWeightsKernel(NeuralNet* nn, double** outputs, 
		double** error, double*** deltaWeights, int layer){

	for(int neuron1 = threadIdx.x + (blockDim.x* blockIdx.x); neuron1 < nn -> neurons[layer]; neuron1 += (blockDim.x * gridDim.x)){
		for(int neuron2 = threadIdx.y + (blockDim.y * blockIdx.y); neuron2 < nn->neurons[layer + 1]; neuron2 += (blockDim.y * gridDim.y)){

			switch(nn->activations[layer][neuron2]){
				case LINEAR:
					deltaWeights[layer][neuron1][neuron2] += 
						(LEARNING_RATE * 
						 error[layer + 1][neuron2] * 
						 outputs[layer][neuron1]);
					break;
		
				case BINARY_STEP:
					// This has a derivative of 0 everywhere so 
					// nothing needs to be added
					break;
		
				case LOGISTIC:
					deltaWeights[layer][neuron1][neuron2] +=
						(LEARNING_RATE * error[layer + 1][neuron2] *
						 outputs[layer][neuron1] * 
						 (outputs[layer][neuron2] * 
						  (1 - outputs[layer][neuron2])));
					break;
		
				case TANH:
					deltaWeights[layer][neuron1][neuron2] +=
						(LEARNING_RATE * error[layer + 1][neuron2] *
						 outputs[layer][neuron1] * 
						 (1 - (outputs[layer][neuron2] * 
						       outputs[layer][neuron2])));
					break;
		
				case RELU:
					if(outputs[layer][neuron2] > 0){
						deltaWeights[layer][neuron1][neuron2] +=
							(LEARNING_RATE * 
							 error[layer + 1][neuron2] * 
							 outputs[layer][neuron1]);
					}
					break;
		
				case LEAKYRELU:
					if(outputs[layer + 1][neuron2] < 0){
						deltaWeights[layer][neuron1][neuron2] +=
							(LEARNING_RATE * 
							 error[layer + 1][neuron2] * 
							 outputs[layer][neuron1] * .01);
					}
		
					else{
						deltaWeights[layer][neuron1][neuron2] += 
							(LEARNING_RATE * 
							 error[layer + 1][neuron2] * 
							 outputs[layer][neuron1]);
					}
					break;
			}

		}
	}
}

/**
  *	Backpropogates the biases in the neural net
  *	Parameter nn: the neural network to backpropogate
  *	Parameter layer: the layer that is being backpropogated
  *	Returns: nothing
  */

__global__
void backpropogationBiasesKernel(NeuralNet* nn, double** outputs, double** error, double** deltaBiases, int layer){

	for(int neuron = threadIdx.x + (blockDim.x * blockIdx.x); neuron < nn->neurons[layer]; neuron += (blockDim.x * gridDim.x)){

		switch(nn->activations[layer][neuron]){
			case LINEAR:
				deltaBiases[layer][neuron] += 
					(LEARNING_RATE * 
					 error[layer][neuron]);
				break;
	
			case BINARY_STEP:
				// This has a derivative of 0 everywhere
				// so nothing needs to be added
				break;
	
			case LOGISTIC:
				deltaBiases[layer][neuron] += 
					(LEARNING_RATE * 
					 error[layer][neuron] * 
					 outputs[layer][neuron] * 
					 (1 - outputs[layer][neuron]));
				break;
	
			case TANH:
				deltaBiases[layer][neuron] += 
					(LEARNING_RATE * 
					 error[layer][neuron] * 
					 (1 - (outputs[layer][neuron] * 
					  outputs[layer][neuron])));
				break;
	
			case RELU:
				if(outputs[layer][neuron] > 0){
					deltaBiases[layer][neuron] += 
						(LEARNING_RATE * 
						 error[layer][neuron]);
				}
				break;
	
			case LEAKYRELU:
				if(outputs[layer][neuron] < 0){
					deltaBiases[layer][neuron] += 
						(LEARNING_RATE * 
						 error[layer][neuron] * 
						 .01);
				}
	
				else{
					deltaBiases[layer][neuron] += 
						(LEARNING_RATE * 
						 error[layer][neuron]);
				}
				break;
		}
	}
}

/**
  *	Changes the weights in the neural net
  *	Parameter nn: the neural network to change the weights in
  *	Parameter deltaWeights: the matrix of total unaveraged weight changes
  *	Parameter numOutputs: the number of outputs to average
  *	Returns: nothing
  */

__global__
void changeWeights(NeuralNet* nn, double*** deltaWeights, int numOutputs){
	for(int layer = threadIdx.x + (blockDim.x * blockIdx.x); layer < nn->layers-1; layer += (blockDim.x * gridDim.x)){
		for(int neuron1 = threadIdx.y + (blockDim.y * blockIdx.y); neuron1 < nn->neurons[layer]; neuron1 += (blockDim.y * gridDim.y)){
			for(int neuron2 = threadIdx.z + (blockDim.z * blockIdx.z); neuron2 < nn->neurons[layer + 1]; neuron2 += (blockDim.z * gridDim.z)){
				nn->weights[layer][neuron1][neuron2] -= (deltaWeights[layer][neuron1][neuron2] / numOutputs);
			}
		}
	}
}

/**
  *	Changes the biases in the neural net
  *	Parameter nn: the neural network to change the biases in
  *	Parameter deltaBiases: the matrix of total unaveraged bias changes
  *	Parameter numOutputs: the total number of outputs to average
  *	Returns: nothing
  */

__global__
void changeBiases(NeuralNet* nn, double** deltaBiases, int numOutputs){
	for(int layer = threadIdx.x + (blockDim.x * blockIdx.x); layer < nn->layers - 1; layer += (blockDim.x * gridDim.x)){
		for(int neuron = threadIdx.y + (blockDim.y * blockIdx.y); neuron < nn->neurons[layer]; neuron += blockDim.y * gridDim.y){
			nn->biases[layer][neuron] -= (deltaBiases[layer][neuron] / numOutputs);
		}
	}
}

/**
  *	Backpropogates the neural network with the actual and expected outputs
  *	Parameter nn: the neural network to backpropogate
  *	Parameter actual: the actual outputs given by the neural network
  *	Parameter expected: the expected outputs given by the neural network
  *	Parameter numOutputs: the number of outputs
  *	Returns: nothing
  */

void backpropogate(NeuralNet* nn, double*** outputs, double** expected, int numOutputs){

	// Gets an matrix for the error
	double** error = getNeuralMatrix(nn);

	double** deltaBiasMatrix=getNeuralMatrix(nn);

	double*** deltaWeightMatrix=getNeuralWeightMatrix(nn);

	// Loops through the outputs
	for(int output = 0; output < numOutputs; output++){
		for(int layer=nn->layers - 1; layer >= 0; layer--){

			// Gets the error
			getErrorKernel<<<NUMBLOCKS, BLOCKSIZE>>>(nn, error, outputs[output], expected[output], layer);
			hipDeviceSynchronize();

			// Backpropgate the weights/biases
			if(layer != nn->layers - 1){
				backpropogationWeightsKernel<<<dim3(NUMBLOCKS, NUMBLOCKS), dim3(BLOCKSIZE/4, BLOCKSIZE/4)>>>(nn, outputs[output], error, deltaWeightMatrix, layer);
				hipDeviceSynchronize();

				backpropogationBiasesKernel<<<NUMBLOCKS, BLOCKSIZE>>>(nn, outputs[output], error, deltaBiasMatrix, layer);
				hipDeviceSynchronize();
			}
		}
	}

	// Changes the weights in the neural net
	changeWeights<<<dim3(NUMBLOCKS, NUMBLOCKS, NUMBLOCKS), dim3(BLOCKSIZE/16, BLOCKSIZE/16, BLOCKSIZE/16)>>>(nn, deltaWeightMatrix, numOutputs);
	hipDeviceSynchronize();

	// Changes the biases in the neural net
	changeBiases<<<dim3(NUMBLOCKS, NUMBLOCKS), dim3(BLOCKSIZE/4, BLOCKSIZE/4)>>>(nn, deltaBiasMatrix, numOutputs);
	hipDeviceSynchronize();

	// Frees the error matrix
	freeNeuralMatrix(error, nn);

	// Frees the bias matrix
	freeNeuralMatrix(deltaBiasMatrix, nn);

	// Frees the weight matrix
	freeNeuralWeightMatrix(deltaWeightMatrix, nn);
}

/**
  *	Allocates memory for and zeros a matrix of the same size as the neural network's nodes
  *	Parameter nn: the neural network to get the dimensions from
  *	Returns: a matrix of the same size as the neural network's nodes
  */

double** getNeuralMatrix(NeuralNet* nn){
	
	double** matrix;
	hipMallocManaged(&matrix, nn->layers*sizeof(double*));

	for(int layer = 0; layer < nn->layers; layer++){
		hipMallocManaged(&matrix[layer], nn->neurons[layer]*sizeof(double));
		
		for(int neuron = 0; neuron < nn->neurons[layer]; neuron++){
			matrix[layer][neuron] = 0;
		}
	}

	return matrix;
}

/**
  *	Frees the neural matrix
  *	Parameter matrix: the matrix to feree
  *	Parameter nn: the neural network to get the dimensions from
  *	Returns: nothing
  */


void freeNeuralMatrix(double** matrix, NeuralNet* nn){
	
	for(int layer = 0; layer < nn->layers; layer++){
		hipFree(matrix[layer]);
	}

	hipFree(matrix);
}

/**
  *	Allocates memory and zeros a matrix of the same size as the neural network's weight matrix
  *	Parameter nn: the neural network to get the weight matrix from
  *	Returns: a matrix of the same size as the neural network's weight matrix
  */

double*** getNeuralWeightMatrix(NeuralNet* nn){

	double*** matrix;
	hipMallocManaged(&matrix, nn->layers * sizeof(double**));

	for(int layer = 0; layer < nn->layers-1; layer++){

		hipMallocManaged(&matrix[layer], nn->neurons[layer] * sizeof(double*));
		for(int neuron1 = 0; neuron1 < nn->neurons[layer]; neuron1++){

			hipMallocManaged(&matrix[layer][neuron1], nn->neurons[layer+1] * sizeof(double));
			for(int neuron2 = 0; neuron2 < nn->neurons[layer + 1]; neuron2++){
				matrix[layer][neuron1][neuron2] = 0;
			}
		}
	}

	return matrix;
}

/**
  *	Frees the neural weight matrix
  *	Parameter matrix: the neural weight matrix
  *	Parameter nn: the neural network to get the dimensions from
  *	Returns: nothing
  */

void freeNeuralWeightMatrix(double*** matrix, NeuralNet* nn){

	for(int layer = 0; layer < nn->layers - 1; layer++){
		for(int neuron1 = 0; neuron1 < nn->neurons[layer]; neuron1++){
			hipFree(matrix[layer][neuron1]);
		}

		hipFree(matrix[layer]);
	}

	hipFree(matrix);
}

