#include "hip/hip_runtime.h"
#include "Game.cuh"

/**
  *	Gets the index of the highest value in a list
  *	Parameter outputs: the list to look through
  *	Parameter numOutputs: the number of elements in
  *	the outputs list
  *	Returns: the index of the highest value in a list
  */

int getMaxIndex(double* outputs, int numOutputs){
	// Gets the index with the highest
	int maxIndex=0;
	for(int output=0; output<numOutputs; output++){
		if(outputs[maxIndex] <= outputs[output]){
			maxIndex=output;
		}
	}
	return maxIndex;
}

/**
  *	Changes an index into a move vector
  *	Parameter index: the index to change into
  *	a move vector
  *	Returns: a vector corresponding to the move to be
  *	made in the form (oldRow, oldCol, newRow, newCol)
  */

int* parseIndexToMove(int index){

	// Gets the actual move from the max
	int* move=(int*)(calloc(4, sizeof(int)));
	for(int movePart=0; movePart<4; movePart++){
		move[movePart]=index%8;
		index/=8;
	}
	return move;
}

/**
  *	Changes the move into an index (for the output list)
  *	This is the inverse of the parseIndexToMove function
  *	and makes a 1-1 correspondance between the functions
  *	Parameter move: the move vector (oldRow, oldCol, 
  *	newRow, newCol)
  *	Returns: the corresponding index of the move
  */

int parseMoveToIndex(int* move){
	int index=0;
	for(int part=0; part<4; part++){
		index+=move[3-part]*pow(8, 3-part);
	}
	return index;
}

/**
  *	Allows the user to input a move to play against the neural network
  *	Parameter board: the board to make a move on
  *	Parameter color: the player's color (that's making the move)
  *	Returns: the winner's color if there is one -1 otherwise
  */

int makePlayerTurn(Piece** board, int color){
	int winner=-1;
	int madeMove=0;
	char* buffer=(char*)calloc(80, sizeof(char));

	do{
		//Handles user input
		printf("Make a numerical move in the format: [oldRow oldCol newRow newCol]\n");

		int oldRow=-1;
		int oldCol=-1;
		int newRow=-1;
		int newCol=-1;
		
		fgets(buffer, 78, stdin);
		oldRow=buffer[0]-48;
		oldCol=buffer[2]-48;
		newRow=buffer[4]-48;
		newCol=buffer[6]-48;

		// Validates the move and gets a winner if there is one
		if(isValidMove(board, oldRow, oldCol, newRow, newCol, color)){
			if(movePiece(board, oldRow, oldCol, newRow, newCol)==KINGREWARD){
				winner=color;
			}
			madeMove=1;
		}

		// An invalid move was made
		else{
			printf("Invalid move made\n");
		}
	}while(!madeMove);

	free(buffer);

	return winner;
}

/**
  *	Makes the turn and updates the output vectors
  *	Parameter board: the board to make a turn on
  *	Parameter color: the color of the player whose turn it is
  *	Parameter turn: the turn number
  *	Parameter nn: the neural network to use
  *	Parameter inputVector: a vector with enough space to hold
  *	all one-hot encoded data values for the board
  *	Parameter output: the rewards corresponding to the 
  *	output of the neural network
  *	Parameter expected: the values that were rewarded
  *	Parameter chosens: the chosen moves
  *	Returns: whether or not there was a checkmate
  */

int makeTurn(Piece** board, int color, int turn, NeuralNet* nn, 
	double* inputVector, double*** output, double** expected, int* chosens){

	// Gets the expected output vector
	oneHotEncode(board, inputVector);
	feedForward(nn, &output[turn], inputVector);

	// Gets the next move
	int madeMove=0;
	int randomMove=((rand()+0.0)/RAND_MAX)<EXPLORATION;
	
	int winner=-1;

	do{

		// If there should be a random move made
		if(randomMove){

			// Gets the random move
			int random=rand();
			int randomIndex=random%nn->neurons[nn->layers-1];
			int* move=parseIndexToMove(randomIndex);

			// Validates, makes, and gets the reward for a move
			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){

				expected[turn][randomIndex]=movePiece(board, 
					move[0], move[1], move[2], move[3])+TURNDEFICIT;

				chosens[turn]=randomIndex;

				if(expected[turn][randomIndex]==KINGREWARD+TURNDEFICIT){
					winner=color;
				}

				madeMove=1;
			}

			// Penalizes invalid moves
			else{
				expected[turn][randomIndex]=-1;
			}

			free(move);

		}
		
		// Otherwise choose the best move
		else{

			int maxIndex=getMaxIndex(output[turn][nn->layers-1], 
					nn->neurons[nn->layers-1]);
			int* move=parseIndexToMove(maxIndex);
			
			// Verifies the move and gets the reward for the move
			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){
				
				// Updates the reward
				expected[turn][maxIndex]=
					movePiece(board, move[0], move[1], 
					move[2], move[3])+TURNDEFICIT;
				
				// Updates the chosen list
				chosens[turn]=maxIndex;

				if(expected[turn][maxIndex]==KINGREWARD+TURNDEFICIT){
					winner=color;
				}

				madeMove=1;
			}

			// Penalizes invalid moves
			else{
				expected[turn][maxIndex]=-1;
			}

			free(move);

			// Make a random move if the best move is invalid
			randomMove=1;	
		}

	}while(!madeMove);

	return winner;
}

/**
  *	Plays a single game
  *	Parameter nn1: the first neural network
  *	Parameter nn2: the second neural network
  *	Parameter playerColor: the color of the player (-1
  *	if no player)
  *	Parameter inputVector: the input vector allocated
  *	with enough room for the one-hot encoded values of
  *	the board
  *	Parameter output1: the output values for the first 
  *	neural network
  *	Parameter output2: the output values for the second
  *	neural network
  *	Parameter expected1: the reward values for the 
  *	first player
  *	Parameter expected2: the reward values for the 
  *	second player
  *	Parameter chosens1: the chosen move indices for the 
  *	first player
  *	Parameter chosens2: the chosen move indices for the
  *	second player
  *	Parameter whiteTurns: the number of turns for the 
  *	first player
  *	Parameter blackTurns: the number of turns for the 
  *	second player
  *	Returns: the winner of the game (-1 if tie)
  */

int playGame(NeuralNet* nn1, NeuralNet* nn2, int playerColor, double* inputVector, 
	double*** output1, double*** output2, double** expected1, 
	double** expected2, int* chosens1, int* chosens2, int* whiteTurns, 
	int* blackTurns){

	Piece** board=makeChessBoard();

	int winner;
	int color=0;
	do{

		// Makes the player's turn
		if(color == playerColor){
			printChessBoard(board);
			winner=makePlayerTurn(board, color);
		}

		// Makes white's turn
		else if(color==0){
			//printf("White %d\n", *whiteTurns);
			winner=makeTurn(board, color, *whiteTurns,  nn1, 
				inputVector, output1, expected1, chosens1);
			(*whiteTurns)++;
		}

		// Makes black's turn
		else{
			//printf("Black %d\n", *blackTurns);
			winner=makeTurn(board, color, *blackTurns, nn2, 
				inputVector, output2, expected2, chosens2);
			(*blackTurns)++;
		}

		color=(color+1)%2;
	}while(winner<0 && (*whiteTurns)<TURNS && (*blackTurns)<TURNS);

	printChessBoard(board);

	freeChessBoard(board);

	return winner;
}

/**
  *	Uses the bellman equation to alter the actual reward values that were 
  *	returned
  *	Parameter nn: the neural network of the corresponding player
  *	Parameter outputs: the outputs of the neural networks
  *	Parameter expected: the reward values
  *	Parameter chosens: the states that were chosen
  *	Parameter numOutputs: the max number of outputs that could be chosen
  *	Parameter won: whether or not the player corresponding to the reward
  *	values has won
  *	Parameter tie: whether or not the player corresponding to the reward
  *	values has tied
  *	Returns: nothing
  */

void alterExpected(NeuralNet* nn, double*** outputs, double** expected, 
	int* chosens, int numOutputs, int won, int tie){
	
	// The bellman equation to chain the actions together
	for(int output=numOutputs-1; output>=0; output--){
		
		// Changes the non-chosen values to the output from the neural network
		// so no erroneous changes are made
		for(int reward=0; reward<nn->neurons[nn->layers-1]; reward++){
			if(expected[output][reward]!=-1 && reward != chosens[output]){
				expected[output][reward]=outputs[output][nn->layers-1][reward];
			}
		}

		// Bellman equation on last state's chosen value
		if(output==numOutputs-1){
			if(won){
				expected[output][chosens[output]]+=DISCOUNT*WINREWARD;
			}
			else if(!tie){
				expected[output][chosens[output]]+=DISCOUNT*LOSSREWARD;
			}
		}

		// Bellman equation on every other state's chosen value
		else{
			expected[output][chosens[output]]+=
				DISCOUNT*expected[output+1][chosens[output+1]];
			chosens[output+1]=-1;
		}
	}
	chosens[0]=-1;
}

/**
  *	Trains the neural network
  *	Parameter nn1: the first neural network to train
  *	Parameter nn2: the second neural network to train
  *	Parameter playerColor: the color of the player (or 
  *	-1 if there is no player)
  *	Returns: nothing
  */

void train(NeuralNet* nn1, NeuralNet* nn2, int playerColor, char* file1, char* file2){
	// The inputs used for both neural network feedforwards
	double* sharedInputs=(double*)calloc(nn1 -> neurons[0], 
		sizeof(double*));

	// Gets the output Matrices for both sides of the neural nets
	double*** output1=makeExpected(nn1, TURNS);
	double*** output2=makeExpected(nn2, TURNS);

	// Gets the reward Matrices for both sides of the neural nets
	double** expected1=makeActual(nn1, TURNS);
	double** expected2=makeActual(nn2, TURNS);

	// Gets the chosens vectors denoting what move was chosen
	int* chosens1=(int*)calloc(TURNS, sizeof(int));
	int* chosens2=(int*)calloc(TURNS, sizeof(int));
	for(int output=0; output<TURNS; output++){
		chosens1[output]=-1;
		chosens2[output]=-1;
	}

	// Loops through infinite games and plays the game to train it
	for(int game=0; 1; game++){

		// Serializes the neural networks every 5 games
		if(game%5==0){
			printf("Serializing the neural networks\n");
			
			serializeNeuralNet(nn1, file1);
			serializeNeuralNet(nn2, file2);
		}

		int* whiteTurns=(int*)calloc(1, sizeof(int));
		int* blackTurns=(int*)calloc(1, sizeof(int));

		// Plays a game
		printf("Training on game %d\n", game);
		int winner=playGame(nn1, nn2, playerColor, sharedInputs, output1, output2, 
			expected1, expected2, chosens1, chosens2, whiteTurns, blackTurns);

		// Changes the rewards using the bellman equation
		alterExpected(nn1, output1, expected1, chosens1, *whiteTurns, winner==0, winner==-1);
		alterExpected(nn2, output2, expected2, chosens2, *blackTurns, winner==1, winner==-1);

		free(whiteTurns);
		free(blackTurns);

		// Backpropogates when there is no player
		if(playerColor==-1){
			printf("Backpropogating\n");
			backpropogate(nn1, output1, expected1, TURNS);
			backpropogate(nn2, output2, expected2, TURNS);
		}
	}
}
