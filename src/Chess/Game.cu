#include "hip/hip_runtime.h"
#include "Game.cuh"

int getMaxIndex(double* outputs, int numOutputs){
	// Gets the index with the highest
	int maxIndex=0;
	for(int output=0; output<numOutputs; output++){
		if(outputs[maxIndex] <= outputs[output]){
			maxIndex=output;
		}
	}
	return maxIndex;
}

int* parseIndexToMove(int index){

	// Gets the actual move from the max
	int* move=(int*)(calloc(4, sizeof(int)));
	for(int movePart=0; movePart<4; movePart++){
		move[movePart]=index%8;
		index/=8;
	}
	return move;
}

int parseMoveToIndex(int* move){
	int index=0;
	for(int part=0; part<4; part++){
		index+=move[3-part]*pow(8, 3-part);
	}
	return index;
}

/**
  *	Makes the turn and updates the output vector
  *	Parameter nn: the neural network
  *	Parameter board: the board to change
  *	Parameter inputVector: the input vector
  *	Parameter outputs: the output matrix to alter for backpropogation
  *	Parameter actual:
  *	Parameter numOutputs: the output matrix
  *	Parameter color: the color of the person to make a move
  *	Returns: whether or not there was a checkmate
  */

int makeTurn(Piece** board, int color, int turn, NeuralNet* nn, 
	double* inputVector, double*** expected, double** actual, int* chosens){
	
	// Gets the expected output vector
	oneHotEncode(board, inputVector);
	feedForward(nn, &expected[turn], inputVector);

	// Gets the next move
	int madeMove=0;
	int randomMove=rand()/RAND_MAX<EXPLORATION;
	
	int winner=-1;

	do{

		// If there should be a random move made
		if(randomMove){

			// Gets the random move
			int random=rand();
			//printf("Random=%d mod neurons=%d=%d\n", random, nn->neurons[nn->layers-1], random%nn->neurons[nn->layers-1]);
			int randomIndex=random%nn->neurons[nn->layers-1];
			int* move=parseIndexToMove(randomIndex);

			// Validates, makes, and gets the reward for a move
			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){

				//printf("Validation suceeded\n");

				actual[turn][randomIndex]=movePiece(board, 
					move[0], move[1], move[2], move[3]);

				//printf("Moved\n");
				chosens[turn]=randomIndex;

				if(actual[turn][randomIndex]==KINGREWARD){
					winner=color;
				}

				madeMove=1;
			}

			else{
				//printf("Validation failed actual[%d][%d]=-1\n", turn, randomIndex);
				actual[turn][randomIndex]=-1;
				//printf("Reward updated\n");
			}

			free(move);

		}
		
		// Otherwise choose the best move
		else{
			int maxIndex=getMaxIndex(expected[turn][nn->layers-1], 
					nn->neurons[nn->layers-1]);
			int* move=parseIndexToMove(maxIndex);

			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){
				//printf("Validation suceeded\n");

				actual[turn][maxIndex]=
					movePiece(board, move[0], move[1], 
					move[2], move[3]);
				
				chosens[turn]=maxIndex;

				if(actual[turn][maxIndex]==KINGREWARD){
					winner=color;
				}

				madeMove=1;
			}

			else{
				//printf("Validation failed actual[%d][%d]=-1\n", turn, maxIndex);
				actual[turn][maxIndex]=-1;
			}

			free(move);

			// Make a random move if the best move is invalid
			randomMove=1;	
		}

	}while(!madeMove);
	
	//printf("MTF\n");

	return winner;
}

/**
  *	Plays a single game
  *	Parameter nn1: the first neural network
  *	Parameter nn2: the second neural network
  *	Parameter inputVector: the input vector
  */

int playGame(NeuralNet* nn1, NeuralNet* nn2, double* inputVector, 
	double*** expected1, double*** expected2, double** actual1, 
	double** actual2, int* chosens1, int* chosens2){

	Piece** board=makeChessBoard();
	//printChessBoard(board);

	int winner;
	int whiteTurns=0;
	int blackTurns=0;
	int color=0;
	do{
		if(color==0){
			winner=makeTurn(board, color, whiteTurns,  nn1, 
				inputVector, expected1, actual1, chosens1);
			whiteTurns++;
		}
		else{
			winner=makeTurn(board, color, blackTurns, nn2, 
				inputVector, expected2, actual2, chosens2);
			blackTurns++;
		}
		color=(color+1)%2;

		//printChessBoard(board);
	}while(winner<0 && whiteTurns<TURNS && blackTurns<TURNS);

	printChessBoard(board);

	return winner;
}

/**
  *	Uses the bellman equation to alter the actual reward values that were 
  *	returned
  *	Parameter actual: the actual reward values
  *	Parameter chosen: the states that were chosen
  *	Parameter numOutputs: the max number of outputs that could be chosen
  *	Returns: nothing
  */

void alterActual(double** actual, int* chosens, int numOutputs){
	for(int output=numOutputs-2; output>=0; output--){
		if(chosens[output+1]!=-1){
			actual[output][chosens[output]]+=
				DISCOUNT*actual[output+1][chosens[output+1]];
		}
		chosens[output+1]=-1;
	}
	chosens[0]=-1;
}

/**
  *	Trains the neural network
  *	Parameter nn1: the first neural network to train
  *	Parameter nn2: the second neural network to train
  *	Returns: nothing
  */

void train(NeuralNet* nn1, NeuralNet* nn2){
	// The inputs used for both neural network feedforwards
	double* sharedInputs=(double*)calloc(nn1 -> neurons[0], 
		sizeof(double*));

	// Gets the expected output Matrices for both sides of the neural nets
	double*** expected1=makeExpected(nn1, TURNS);
	double*** expected2=makeExpected(nn2, TURNS);

	// Gets the actual output Matrices for both sides of the neural nets
	double** actual1=makeActual(nn1, TURNS);
	double** actual2=makeActual(nn2, TURNS);

	// Gets the chosens vectors denoting what move was chosen
	int* chosens1=(int*)calloc(TURNS, sizeof(int));
	int* chosens2=(int*)calloc(TURNS, sizeof(int));
	for(int output=0; output<TURNS; output++){
		chosens1[output]=-1;
		chosens2[output]=-1;
	}

	// Loops through infinite games and plays the game to train it
	for(int game=0; 1; game++){
		if(game%5==0){
			printf("Serializing the neural networks\n");
			serializeNeuralNet(nn1, "nn1.txt");
			serializeNeuralNet(nn2, "nn2.txt");
		}

		printf("Training on game %d\n", game);
		playGame(nn1, nn2, sharedInputs, expected1, expected2, 
			actual1, actual2, chosens1, chosens2);
		
		alterActual(actual1, chosens1, TURNS);
		alterActual(actual2, chosens2, TURNS);

		printf("Backpropogating\n");

		backpropogate(nn1, expected1, actual1, TURNS);
		backpropogate(nn2, expected2, actual2, TURNS);
	}
}
