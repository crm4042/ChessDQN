#include "hip/hip_runtime.h"
#include "Game.cuh"

int getMaxIndex(double* outputs, int numOutputs){
	// Gets the index with the highest
	int maxIndex=0;
	for(int output=0; output<numOutputs; output++){
		if(outputs[maxIndex] <= outputs[output]){
			maxIndex=output;
		}
	}
	return maxIndex;
}

int* parseIndexToMove(int index){

	// Gets the actual move from the max
	int* move=(int*)(calloc(4, sizeof(int)));
	for(int movePart=0; movePart<4; movePart++){
		move[movePart]=index%8;
		index/=8;
	}
	return move;
}

int parseMoveToIndex(int* move){
	int index=0;
	for(int part=0; part<4; part++){
		index+=move[3-part]*pow(8, 3-part);
	}
	return index;
}

int makePlayerTurn(Piece** board, int color){
	int winner=-1;
	int madeMove=0;
	char* buffer=(char*)calloc(80, sizeof(char));

	do{
		printf("Make a move in the format: [oldRow oldCol newRow newCol]\n");

		int oldRow=-1;
		int oldCol=-1;
		int newRow=-1;
		int newCol=-1;
		
		fgets(buffer, 78, stdin);
		oldRow=buffer[0]-48;
		oldCol=buffer[2]-48;
		newRow=buffer[4]-48;
		newCol=buffer[6]-48;

		if(isValidMove(board, oldRow, oldCol, newRow, newCol, color)){
			if(movePiece(board, oldRow, oldCol, newRow, newCol)==KINGREWARD){
				winner=color;
			}
			madeMove=1;
		}
		else{
			printf("Invalid move made\n");
		}
	}while(!madeMove);

	free(buffer);

	return winner;
}

/**
  *	Makes the turn and updates the output vector
  *	Parameter nn: the neural network
  *	Parameter board: the board to change
  *	Parameter inputVector: the input vector
  *	Parameter outputs: the output matrix to alter for backpropogation
  *	Parameter actual:
  *	Parameter numOutputs: the output matrix
  *	Parameter color: the color of the person to make a move
  *	Returns: whether or not there was a checkmate
  */

int makeTurn(Piece** board, int color, int turn, NeuralNet* nn, 
	double* inputVector, double*** expected, double** actual, int* chosens){

	// Gets the expected output vector
	oneHotEncode(board, inputVector);
	feedForward(nn, &expected[turn], inputVector);

	// Gets the next move
	int madeMove=0;
	int randomMove=((rand()+0.0)/RAND_MAX)<EXPLORATION;
	
	int winner=-1;

	do{

		// If there should be a random move made
		if(randomMove){

			//printf("Random move\n");

			// Gets the random move
			int random=rand();
			int randomIndex=random%nn->neurons[nn->layers-1];
			int* move=parseIndexToMove(randomIndex);

			//printf("Move %d %d %d %d\n", move[0], move[1], move[2], move[3]);

			// Validates, makes, and gets the reward for a move
			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){

				//printf("Valid\n");

				// Updates the reward
				actual[turn][randomIndex]=movePiece(board, 
					move[0], move[1], move[2], move[3])+TURNDEFICIT;

				// Gets the index of the chosen value
				chosens[turn]=randomIndex;

				if(actual[turn][randomIndex]==KINGREWARD+TURNDEFICIT){
					winner=color;
				}

				madeMove=1;
			}

			// An invalid move
			else{
				//printf("Not valid\n");
				actual[turn][randomIndex]=-1;
			}

			free(move);

		}
		
		// Otherwise choose the best move
		else{
			//printf("Not random\n");

			int maxIndex=getMaxIndex(expected[turn][nn->layers-1], 
					nn->neurons[nn->layers-1]);
			int* move=parseIndexToMove(maxIndex);
			
			//printf("Move %d %d %d %d\n", move[0], move[1], move[2], move[3]);

			if(isValidMove(board, move[0], move[1], move[2], 
						move[3], color)){

				//printf("Valid");
				
				// Updates the reward
				actual[turn][maxIndex]=
					movePiece(board, move[0], move[1], 
					move[2], move[3])+TURNDEFICIT;
				
				// Updates the chosen list
				chosens[turn]=maxIndex;

				if(actual[turn][maxIndex]==KINGREWARD+TURNDEFICIT){
					winner=color;
				}

				madeMove=1;
			}

			else{
				//printf("Not valid");

				actual[turn][maxIndex]=-1;
			}

			free(move);

			// Make a random move if the best move is invalid
			randomMove=1;	
		}

	}while(!madeMove);

	return winner;
}

/**
  *	Plays a single game
  *	Parameter nn1: the first neural network
  *	Parameter nn2: the second neural network
  *	Parameter inputVector: the input vector
  */

int playGame(NeuralNet* nn1, NeuralNet* nn2, int playerColor, double* inputVector, 
	double*** expected1, double*** expected2, double** actual1, 
	double** actual2, int* chosens1, int* chosens2, int* whiteTurns, 
	int* blackTurns){

	Piece** board=makeChessBoard();
	//printChessBoard(board);

	int winner;
	int color=0;
	do{
		char* filename=(char*)calloc(10, sizeof(char));
		strcpy(filename, "board.txt\0");
		serializeChessBoard(board, filename);
		free(filename);
		if(color == playerColor){
			printChessBoard(board);
			winner=makePlayerTurn(board, color);
		}
		else if(color==0){
			//printf("\nWhite turn %d\n", *whiteTurns);
			winner=makeTurn(board, color, *whiteTurns,  nn1, 
				inputVector, expected1, actual1, chosens1);
			(*whiteTurns)++;
		}
		else{
			//printf("\nBlack turn %d\n", *blackTurns);
			winner=makeTurn(board, color, *blackTurns, nn2, 
				inputVector, expected2, actual2, chosens2);
			(*blackTurns)++;
		}
		color=(color+1)%2;

		//printChessBoard(board);
	}while(winner<0 && (*whiteTurns)<TURNS && (*blackTurns)<TURNS);

	printChessBoard(board);

	freeChessBoard(board);

	return winner;
}

/**
  *	Uses the bellman equation to alter the actual reward values that were 
  *	returned
  *	Parameter actual: the actual reward values
  *	Parameter chosen: the states that were chosen
  *	Parameter numOutputs: the max number of outputs that could be chosen
  *	Returns: nothing
  */

void alterActual(double** actual, int* chosens, int numOutputs, int won, int tie){
	// The win/loss rewards to be added to the final state
	if(won){
		actual[numOutputs-1][chosens[numOutputs-1]]+=DISCOUNT*WINREWARD;
	}
	else if(!tie){
		actual[numOutputs-1][chosens[numOutputs-1]]+=DISCOUNT*LOSSREWARD;
	}

	// The bellman equation to chain the actions together
	for(int output=numOutputs-2; output>=0; output--){
		if(chosens[output+1]!=-1){
			actual[output][chosens[output]]+=
				DISCOUNT*actual[output+1][chosens[output+1]];
		}
		chosens[output+1]=-1;
	}
	chosens[0]=-1;
}

/**
  *	Trains the neural network
  *	Parameter nn1: the first neural network to train
  *	Parameter nn2: the second neural network to train
  *	Returns: nothing
  */

void train(NeuralNet* nn1, NeuralNet* nn2, int playerColor){
	// The inputs used for both neural network feedforwards
	double* sharedInputs=(double*)calloc(nn1 -> neurons[0], 
		sizeof(double*));

	// Gets the expected output Matrices for both sides of the neural nets
	double*** expected1=makeExpected(nn1, TURNS);
	double*** expected2=makeExpected(nn2, TURNS);

	// Gets the actual output Matrices for both sides of the neural nets
	double** actual1=makeActual(nn1, TURNS);
	double** actual2=makeActual(nn2, TURNS);

	// Gets the chosens vectors denoting what move was chosen
	int* chosens1=(int*)calloc(TURNS, sizeof(int));
	int* chosens2=(int*)calloc(TURNS, sizeof(int));
	for(int output=0; output<TURNS; output++){
		chosens1[output]=-1;
		chosens2[output]=-1;
	}

	// Loops through infinite games and plays the game to train it
	for(int game=0; 1; game++){
		if(game%5==0){
			printf("Serializing the neural networks\n");
			char* buffer=(char*)calloc(8, sizeof(char));
			
			strcpy(buffer, "nn1.txt\0");
			serializeNeuralNet(nn1, buffer);
			
			strcpy(buffer, "nn2.txt\0");
			serializeNeuralNet(nn2, buffer);
			
			free(buffer);
		}

		int* whiteTurns=(int*)calloc(1, sizeof(int));
		int* blackTurns=(int*)calloc(1, sizeof(int));

		printf("Training on game %d\n", game);
		int winner=playGame(nn1, nn2, playerColor, sharedInputs, expected1, expected2, 
			actual1, actual2, chosens1, chosens2, whiteTurns, blackTurns);
		
		printf("Performing bellman equation on rewards\n");

		alterActual(actual1, chosens1, *whiteTurns, winner==0, winner==-1);
		alterActual(actual2, chosens2, *blackTurns, winner==1, winner==-1);

		free(whiteTurns);
		free(blackTurns);

		if(playerColor==-1){
			printf("Backpropogating\n");
			backpropogate(nn1, expected1, actual1, TURNS);
			backpropogate(nn2, expected2, actual2, TURNS);
		}
	}
}
