#include "hip/hip_runtime.h"
#include "VerifyMove.cuh"

int getCheck(Piece** board, int kingRow, int kingCol, int color){

	// Loops through the rows
	for(int direction=-1; direction<=1; direction+=2){
		for(int row=kingRow; row<DIM && row>=0; row+=direction){
			if(board[row][kingCol].piece.color==color){
				break;
			}

			else if(board[row][kingCol].piece.isRook || 
				board[row][kingCol].piece.isQueen || 
				(board[row][kingCol].piece.isKing && 
					abs(row-kingRow)==1)){
				return 1;
			}
		}
	}

	// Loops through the cols
	for(int direction=-1; direction<=1; direction+=2){
		for(int col=kingCol; col<DIM && col>=0; col+=direction){
			if(board[kingRow][col].piece.color==color){
				break;
			}

			else if(board[kingRow][col].piece.isRook ||
				board[kingRow][col].piece.isQueen || 
				(board[kingRow][col].piece.isKing && 
					abs(col-kingCol)==1)){
				return 1;
			}
		}
	}

	// Loops through the diagonals
	for(int rowDirection=-1; rowDirection<=1; rowDirection+=2){
		for(int colDirection=-1; colDirection<=1; colDirection+=2){
			for(int row=kingRow, col=kingCol; 
				row>=0 && row<DIM && col>=0 && col<DIM;
				row+=rowDirection, col+=colDirection){

				if(board[row][col].piece.color==color){
					break;
				}

				else if(board[row][col].piece.isBishop || 
					board[row][col].piece.isQueen ||
					(board[row][col].piece.isKing && 
						abs(row-kingRow)==1 && 
						abs(col-kingCol)==1)){

					return 1;
				}
			}
		}
	}

	// Loops through the knights
	for(int row=max(0, kingRow-2); row<=min(DIM-1, kingRow+2); row++){
		for(int col=max(0, kingCol-2); col<=min(DIM-1, kingCol+2); col++){
			if(row==kingRow || col==kingCol || 
				abs(row-kingRow)==abs(col-kingCol)){
				continue;
			}

			else if(board[row][col].piece.isKnight){
				return 1;
			}
		}
	}

	return 0;
}

/**
  *	Gets the change in row
  *	Parameter oldRow: the row to move from
  *	Parameter newRow: the row to move to
  *	Returns: the change in row
  */

int getDeltaRow(int oldRow, int newRow){
	return newRow-oldRow;
}

/**
  *	Gets the change in col
  *	Parameter oldCol: the col to move from
  *	Parameter newCol: the col to move to
  *	Returns: the change in col
  */

int getDeltaCol(int oldCol, int newCol){
	return newCol-oldCol;
}


/**
  *	Checks to make sure the pawn move was valid
  *	Parameter board: the board to check the move from
  *	Parameter oldRow: the row that the pawn moves from
  *	Parameter oldCol: the col that the pawn moves from
  *	Parameter newRow: the row that the pawn moves to
  *	Parameter newCol: the col that the pawn moves to
  *	Parameter color: the color of the pawn to move
  *	Returns: whether or not the move was valid
  */

int isValidPawnMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){
	
	int deltaRow=getDeltaRow(oldRow, newRow);
	int deltaCol=getDeltaCol(oldCol, newCol);
	int colorFactor=pow(-1, color);

	// Double square advance
	if(deltaRow==2*colorFactor && 
			deltaCol==0 &&
			board[oldRow][oldCol].piece.isFirstMove==1){
		
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 0, 0);
	}

	// Single square advance
	else if(deltaRow==1*colorFactor && deltaCol==0){
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 0, 0);
	}

	// Take
	else if(deltaRow==1*colorFactor && abs(deltaCol)==1 && 
			hasEnemy(board, newRow, newCol, color)){
		//printf("%d \n", board[newRow][newCol].numberConversion);
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 1, 0);
	}

	// Invalid move
	else{
		return 0;
	}
}

/**
  *	Checks to make sure the rook move was valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the rook moves from
  *	Parameter oldCol: the col that the rook moves from
  *	Parameter newRow: the row that the rook moves to
  *	Parameter newCol: the col that the rook moves to
  *	Parameter color: the color of the rook to move
  *	Returns: whether or not the move was valid
  */

int isValidRookMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){
	
	int deltaRow=getDeltaRow(oldRow, newRow);
	int deltaCol=getDeltaCol(oldCol, newCol);
	
	// If it matches the parrern for a rook's move
	if((deltaRow!=0 && deltaCol==0) || 
			(deltaRow==0 && deltaCol!=0)){
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 1, 0);
	}

	// Invalid move pattern
	else{
		return 0;
	}
}

/**
  *	Checks to make sure that the knight move was valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the knight moves from
  *	Parameter oldCol: the col that the knight moves from
  *	Parameter newRow: the row that the knight moves from
  *	Parameter newCol: the col that the knight moves from
  *	Parameter color: the color of the knight to move
  *	Returns: whether or not the knight move was valid
  */

int isValidKnightMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){

	int deltaRow=getDeltaRow(oldRow, newRow);
	int deltaCol=getDeltaCol(oldCol, newCol);

	// If it matches the pattern for a knight's move
	if((abs(deltaRow)==1 && abs(deltaCol)==2) || 
			(abs(deltaRow)==2 && abs(deltaCol)==1)){
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 1, 0);
	}

	// Invalid move pattern
	else{
		return 0;
	}
}

/**
  *	Checks to make sure that the bishop's move was valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the bishop moves from
  *	Parameter oldCol: the col that the bishop moves from
  *	Parameter newRow: the row that the bishop moves to
  *	Parameter newCol: the col that the bishop moves to
  *	Parameter color: the color of the bishop to move
  *	Returns: whether or not the bishop's move was valid
  */

int isValidBishopMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){

	int deltaRow=getDeltaRow(oldRow, newRow);
	int deltaCol=getDeltaCol(oldCol, newCol);

	// If it matches the pattern for a bishop's move
	if(abs(deltaRow)==abs(deltaCol)){
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 1, 0);
	}

	// Invalid move pattern
	else{
		return 0;
	}
}

/**
  *	Checks to make sure that the queen's move was valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the queen moves from
  *	Parameter oldCol: the col that the queen moves from
  *	Parameter newRow: the row that the queen moves to
  *	Parameter newCol: the col that the queen moves to
  *	Parameter color: the color of the queen to move
  *	Returns: whether or not the queen's move was valid
  */

int isValidQueenMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){
	return isValidRookMove(board, oldRow, oldCol, newRow, newCol, color) ||
		isValidBishopMove(board, oldRow, oldCol, newRow, newCol, 
				color);
}

/**
  *	Checks to make sure that the king's move was valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the king moves from
  *	Parameter oldCol: the col that the king moves from
  *	Parameter newRow: the row that the king moves to
  *	Parameter newCol: the col that the king moves to
  *	Parameter color: the color of the king to move
  *	Returns: whether or not the king's move was valid
  */

int isValidKingMove(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color){

	int deltaRow=getDeltaRow(oldRow, newRow);
	int deltaCol=getDeltaCol(oldCol, newCol);

	// If it matches the pattern for a king's move
	if((abs(deltaRow)==1 && abs(deltaCol)<=1) || 
			(abs(deltaRow) <=1 && abs(deltaCol)==1)){
		return canMoveFromTo(board, oldRow, oldCol, newRow, newCol, 
				color, 1, 0);
	}

	// If it matches the pattern for a castle
	else if(oldRow==newRow &&
		abs(deltaCol)==2 &&
		board[oldRow][oldCol].piece.isFirstMove &&
		canMoveFromTo(board, oldRow, oldCol, newRow, newCol, color, 
			0, 0) &&
		((deltaCol<0 && board[oldRow][0].piece.isRook && 
			board[oldRow][0].piece.isFirstMove && 
			canMoveFromTo(board, oldRow, 0, newRow, 3, color, 
				0, 0)) ||
		(deltaCol>0 && board[oldRow][7].piece.isRook && 
			board[oldRow][7].piece.isFirstMove && 
			canMoveFromTo(board, oldRow, 7, newRow, 5, color, 
				0, 0)))){

		//Checks for a check between the two (inclusive)
		for(int col=oldCol; col!=newCol+(deltaCol/abs(deltaCol)); col+=(deltaCol/abs(deltaCol))){
			if(getCheck(board, oldRow, col, color)){
				return 0;
			}
		}

		return 1;
	}

	// Invalid move pattern
	else{
		return 0;
	}
}

#include "ChessBoard.cuh"

/**
  *	Checks to see if a move is valid
  *	Parameter board: the board to check the move on
  *	Parameter oldRow: the row that the piece moves from
  *	Parameter oldCol: the col that the piece moves from
  *	Parameter newRow: the row that the piece moves to
  *	Parameter newCol: the col that the piece moves to
  *	Parameter color: the color of the person that's moving
  *	Returns: whether or not the move is valid
  */

int isValidMove(Piece** board, int oldRow, int oldCol, int newRow, int newCol, 
		int color){

	/*printf("OldRow %d, OldCol %d NewRow %d, NewCol %d; ", oldRow, oldCol, newRow, newCol);
	if(verifyBounds(oldRow, oldCol)){
		printf("Valid old bounds ");
		if(verifyBounds(newRow, newCol)){
			printf("Valid new bounds ");
			if(isOccupied(board, oldRow, oldCol)){
				printf("Occupied start position ");
				if(board[oldRow][oldCol].piece.color==color){
					printf("Valid color move ");
					if(!hasObstructions(board, oldRow, oldCol, newRow, newCol)){
						printf("No obstructions. Passed to inside\n");
					}
					else{
						printf("Obstructions found \n");
					}
				}
				else{
					printf("Invalid color move \n");
				}
			}
			else{
				printf("Occupied end position /n");
			}
		}
		else{
			printf("Non-valid new bounds /n");
		}
	}
	else{
		printf("Non-valid old bounds /n");
	}*/

	// Checks if there is a piece at oldRow oldCol that can be moved
	if(verifyBounds(oldRow, oldCol) && verifyBounds(newRow, newCol) && 
			(oldRow!=newRow || oldCol!=newCol) &&
			isOccupied(board, oldRow, oldCol) && 
			board[oldRow][oldCol].piece.color==color){
		
		//printf("Inside\n");

		// Checks each of the pieces
		if(board[oldRow][oldCol].piece.isPawn){
			/*if(newRow==6 && oldCol==newCol){
				printChessBoard(board);
				printf("PAWN oldRow=%d, oldCol=%d, newRow=%d, newCol=%d\n", oldRow, oldCol, newRow, newCol);
			}*/
			return isValidPawnMove(board, oldRow, oldCol, newRow, 
					newCol, color);
		}
		else if(board[oldRow][oldCol].piece.isRook){
			//printf("ROOK\n");
			return isValidRookMove(board, oldRow, oldCol, newRow, 
					newCol, color);
		}
		else if(board[oldRow][oldCol].piece.isKnight){
			//printf("KNIGHT\n");
			return isValidKnightMove(board, oldRow, oldCol, 
					newRow, newCol, color);
		}
		else if(board[oldRow][oldCol].piece.isBishop){
			//printf("BISHOP\n");
			return isValidBishopMove(board, oldRow, oldCol, 
					newRow, newCol, color);
		}
		else if(board[oldRow][oldCol].piece.isQueen){
			//printf("QUEEN\n");
			return isValidQueenMove(board, oldRow, oldCol, 
					newRow, newCol, color);
		}
		else if(board[oldRow][oldCol].piece.isKing){
			//printf("KING\n");
			return isValidKingMove(board, oldRow, oldCol, newRow, 
					newCol, color);
		}
		else{
			printf("");
		}
	}

	return 0;
}

/**
  *	Verifies the bounds of the new position to make sure it is in the board
  *	Parameter row: the row to check the bounds of
  *	Parameter col: the column to check the bounds of
  *	Returns: whether or now newRow and newCol are in the bounds of the board
  */

int verifyBounds(int row, int col){
	return row>=0 && row<DIM && col>=0 && col<DIM;
}

/**
  *	Checks whether or not the square is occupied
  *	Parameter board: the board to check for the occupied square in
  *	Parameter row: the row to check if it is occupied
  *	Parameter col: the col to check if it is occupied
  *	Returns: whether or not the square is occupied
  */

int isOccupied(Piece** board, int row, int col){
	return board[row][col].numberConversion!=0;
}

/**
  *	Checks whether or not the square has an enemy
  *	Parameter board: the board to check for the enemy
  *	Parameter row: the row to check for an enemy
  *	Parameter col: the column to check for an enemy
  *	Parameter color: the color of the side looking for an enemy
  *	Returns: whether or not there is an enemy in row, col
  */

int hasEnemy(Piece** board, int row, int col, int color){
	return board[row][col].numberConversion != 0 && 
		board[row][col].piece.color!=color;
}

/**
  *	Checks if there are obstructions between the start and end
  *	positions
  *	Parameter board: the board to check for obstructions
  *	Parameter oldRow: the row that the piece moves from
  *	Parameter oldCol: the col that the piece moves from
  *	Parameter newRow: the row that the piece moves to
  *	Parameter newCol: the col that the piece moves to
  *	Returns: whether or not there are obstructions between the
  *	start and end positions
  */

int hasObstructions(Piece** board, int oldRow, int oldCol, 
		int newRow, int newCol){
	
	int deltaRow=pow(-1, newRow<oldRow);
	int deltaCol=pow(-1, newCol<oldCol);
	
	// If it moves along the col
	if(oldRow==newRow){
		for(int col=oldCol+deltaCol; col!=newCol; col+=deltaCol){
			if(board[oldRow][col].numberConversion!=0){
				return 1;
			}
		}
	}

	// If it moves along the row
	else if(oldCol==newCol){
		for(int row=oldRow+deltaRow; row!=newRow; row+=deltaRow){
			if(board[row][oldCol].numberConversion!=0){
				return 1;
			}
		}
	}

	// A diagonal
	else if(abs(oldRow-newRow)==abs(oldCol-newCol)){
		for(int row=oldRow+deltaRow, col=oldCol+deltaCol; 
			row!=newRow || col!=newCol; 
			row+=deltaRow, col+=deltaCol){
			if(board[row][col].numberConversion!=0){
				return 1;
			}
		}
	}

	return 0;
}

/**
  *	Checks whether or not the piece can move between the two
  *	locations
  *	Parameter board: the board to check
  *	Parameter oldRow: the old row to check
  *	Parameter oldCol: the old col to check
  *	Parameter newRow: the new row to check
  *	Parameter newCol: the new col to check
  *	Return: whether or not the move can be made
  */

int canMoveFromTo(Piece** board, int oldRow, int oldCol, int newRow, 
		int newCol, int color, int canTake, int canJump){

	return verifyBounds(oldRow, oldCol) && 
		verifyBounds(newRow, newCol) &&
		((canTake && hasEnemy(board, newRow, newCol, color)) ||
		!isOccupied(board, newRow, newCol)) &&
		(canJump || !hasObstructions(board, oldRow, oldCol, 
					     newRow, newCol));
}


