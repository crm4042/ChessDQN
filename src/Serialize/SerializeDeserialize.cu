#include "SerializeDeserialize.cuh"

void serializeNeuralNet(NeuralNet* nn, char* fileName){
	// Opens the file for writing
	FILE* file=fopen(fileName, "w");

	// Writes the layer data
	fprintf(file, "%d\n", nn->layers);

	// Writes the neuron data
	for(int layer=0; layer<nn->layers; layer++){
		fprintf(file, "%d\n", nn->neurons[layer]);
	}

	// Writes the weight data
	for(int layer=0; layer<nn->layers-1; layer++){
		for(int neuron1=0; neuron1<nn->neurons[layer]; neuron1++){
			for(int neuron2=0; neuron2<nn->neurons[layer+1]; neuron2++){
				fprintf(file, "%lf\n", nn->weights[layer][neuron1][neuron2]);
			}
		}
	}

	// Writes the bias data
	for(int layer=0; layer<nn->layers-1; layer++){
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fprintf(file, "%lf\n", nn->biases[layer][neuron]);
		}
	}

	fclose(file);
}

NeuralNet* deserializeNeuralNet(char* fileName){
	FILE* file=fopen(fileName, "r");

	NeuralNet* nn;
	hipMallocManaged(&nn, 1*sizeof(NeuralNet));

	// Gets the layers
	fscanf(file, "%d\n", &nn->layers);

	// Gets the neuron data
	hipMallocManaged(&nn->neurons, nn->layers*sizeof(int));
	for(int layer=0; layer<nn->layers; layer++){
		fscanf(file, "%d\n", &nn->neurons[layer]);
	}

	// Gets the weight data
	hipMallocManaged(&nn->weights, (nn->layers-1)*sizeof(double**));
	for(int layer=0; layer<nn->layers-1; layer++){
		hipMallocManaged(&nn->weights[layer], nn->neurons[layer]*sizeof(double*));
		for(int neuron1=0; neuron1<nn->neurons[layer]; neuron1++){
			hipMallocManaged(&nn->weights[layer][neuron1], 
				nn->neurons[layer+1]*sizeof(double));
			for(int neuron2=0; neuron2<nn->neurons[layer]; neuron2++){
				fscanf(file, "%lf\n", &nn->weights[layer][neuron1][neuron2]);
				printf("Layer=%d\tNeuron1=%d\tNeuron2=%d\tWeight=%lf\n", layer, neuron1, neuron2, nn->weights[layer][neuron1][neuron2]);
			}
		}
	}

	// Gets the bias data
	hipMallocManaged(&nn->biases, (nn->layers-1)*sizeof(double*));
	for(int layer=0; layer<nn->layers-1; layer++){
		hipMallocManaged(&nn->biases[layer], nn->neurons[layer+1]*sizeof(double));
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fscanf(file, "%lf\n", &nn->biases[layer][neuron]);
		}
	}

	fclose(file);

	return nn;
}
