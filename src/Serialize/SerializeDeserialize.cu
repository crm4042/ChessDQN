#include "SerializeDeserialize.cuh"

void serializeNeuralNet(NeuralNet* nn, char* fileName){
	// Opens the file for writing
	FILE* file=fopen(fileName, "w");

	// Writes the layer data
	fprintf(file, "%d\n", nn->layers);

	// Writes the neuron data
	for(int layer=0; layer<nn->layers; layer++){
		fprintf(file, "%d\n", nn->neurons[layer]);
	}

	// Writes the weight data
	for(int layer=0; layer<nn->layers-1; layer++){
		for(int neuron1=0; neuron1<nn->neurons[layer]; neuron1++){
			for(int neuron2=0; neuron2<nn->neurons[layer+1]; neuron2++){
				fprintf(file, "%lf\n", nn->weights[layer][neuron1][neuron2]);
			}
		}
	}

	// Writes the bias data
	for(int layer=0; layer<nn->layers-1; layer++){
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fprintf(file, "%lf\n", nn->biases[layer][neuron]);
		}
	}

	// Writes the activation data
	for(int layer=0; layer<nn->layers-1; layer++){
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fprintf(file, "%d\n", nn->activations[layer][neuron]);
		}
	}

	fclose(file);
}

NeuralNet* deserializeNeuralNet(char* fileName){
	FILE* file=fopen(fileName, "r");

	NeuralNet* nn;
	hipMallocManaged(&nn, 1*sizeof(NeuralNet));

	// Gets the layers
	fscanf(file, "%d\n", &nn->layers);

	// Gets the neuron data
	hipMallocManaged(&nn->neurons, nn->layers*sizeof(int));
	for(int layer=0; layer<nn->layers; layer++){
		fscanf(file, "%d\n", &nn->neurons[layer]);
	}

	// Gets the weight data
	hipMallocManaged(&nn->weights, (nn->layers-1)*sizeof(double**));
	for(int layer=0; layer<nn->layers-1; layer++){
		hipMallocManaged(&nn->weights[layer], nn->neurons[layer]*sizeof(double*));
		for(int neuron1=0; neuron1<nn->neurons[layer]; neuron1++){
			hipMallocManaged(&nn->weights[layer][neuron1], 
				nn->neurons[layer+1]*sizeof(double));
			for(int neuron2=0; neuron2<nn->neurons[layer+1]; neuron2++){
				fscanf(file, "%lf\n", &nn->weights[layer][neuron1][neuron2]);
				//printf("Layer=%d\tNeuron1=%d\tNeuron2=%d\tWeight=%lf\n", layer, neuron1, neuron2, nn->weights[layer][neuron1][neuron2]);
			}
		}
	}

	// Gets the bias data
	hipMallocManaged(&nn->biases, (nn->layers-1)*sizeof(double*));
	for(int layer=0; layer<nn->layers-1; layer++){
		hipMallocManaged(&nn->biases[layer], nn->neurons[layer+1]*sizeof(double));
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fscanf(file, "%lf\n", &nn->biases[layer][neuron]);
		}
	}

	// Gets the activation function data
	hipMallocManaged(&nn->activations, (nn->layers-1)*sizeof(activation*));
	for(int layer=0; layer<nn->layers-1; layer++){
		hipMallocManaged(&nn->activations[layer], nn->neurons[layer+1]*sizeof(activation));
		for(int neuron=0; neuron<nn->neurons[layer+1]; neuron++){
			fscanf(file, "%d\n", &nn->activations[layer][neuron]);
		}
	}

	fclose(file);

	return nn;
}

void serializeChessBoard(Piece** board, char* filename){
	FILE* file=fopen(filename, "w");
	
	fprintf(file, "\t");

	for(int col=0; col<DIM; col++){
		fprintf(file, "%c\t", ((int)'A')+col);
	}

	fprintf(file, "\n");

	for(int row=0; row<DIM; row++){

		fprintf(file, "%d\t", row);

		for(int col=0; col<DIM; col++){
			
			if(board[row][col].numberConversion==0){
				fprintf(file, "______\t");
			}

			else{
				fprintf(file, "__");

				if(board[row][col].piece.color==0){
					fprintf(file, "W");
				}
				else{
					fprintf(file, "B");
				}

				if(board[row][col].piece.isPawn){
					fprintf(file, "P");
				}
				else if(board[row][col].piece.isRook){
					fprintf(file, "R");
				}
				else if(board[row][col].piece.isKnight){
					fprintf(file, "N");
				}
				else if(board[row][col].piece.isBishop){
					fprintf(file, "B");
				}
				else if(board[row][col].piece.isQueen){
					fprintf(file, "Q");
				}
				else{
					fprintf(file, "K");
				}

				fprintf(file, "__\t");
			}
		}

		fprintf(file, "\n");
	}

	fprintf(file, "\n======================================================\n\n");

	for(int row=0; row<DIM; row++){
		for(int col=0; col<DIM; col++){
			fprintf(file, "Row %d, Col %d, Num %d, clr %d, fst %d, Pwn %d, Rk %d, Knt %d, Bshp %d, Qn %d, Kng %d\n", row, col, board[row][col].numberConversion, board[row][col].piece.color, board[row][col].piece.isFirstMove, board[row][col].piece.isPawn, board[row][col].piece.isRook, board[row][col].piece.isKnight, board[row][col].piece.isBishop, board[row][col].piece.isQueen, board[row][col].piece.isKing);
		}
	}

	fclose(file);
}
